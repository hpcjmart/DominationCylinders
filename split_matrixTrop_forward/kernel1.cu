#include <stdio.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"
#include "common.h"

int num_submatrix = 1;


void msplitm(unsigned long m, 
	     unsigned long n, 
	     unsigned long k, 
	     const short *A, 
	     unsigned long lda, 
	     const short *B, 
	     unsigned long ldb, 
	     short *C, 
	     unsigned long ldc)
{

	//float alfa=1.0;
	//float beta=0.0;

    unsigned long  A_sz = m * k;
    unsigned long  B_sz = n * k;
    unsigned long  MAX =  (unsigned long long )m* (unsigned long long) n / num_submatrix;

	MAX -= MAX % k;
	unsigned long numSubMatrixB;
	if (MAX==0) numSubMatrixB = 1;
        else numSubMatrixB = B_sz / MAX;
	unsigned long subCols = B_sz / (numSubMatrixB * k);


	unsigned long numSubMatrixA;
	if (MAX==0) numSubMatrixA = 1;
        else numSubMatrixA = A_sz / MAX;
	unsigned long subRows = A_sz / (numSubMatrixA * k);
	
	unsigned long overflowA = m % subRows;
	unsigned long overflowB = n % subCols;


	for(unsigned long i = 0; i < numSubMatrixB + 1; ++i){
		if(overflowB == 0 && i == numSubMatrixB){
			break;
		}

		short *b;
		short *temp3 = (short*) malloc( sizeof(short)*subCols * k );
		for(unsigned long j = 0; j < k; ++j){
			for(unsigned long x = 0; x < subCols; ++x){
				if(i * subCols + x < n){
					temp3[j * subCols + x] = B[j * n + (i*subCols + x)];
				}else{
					temp3[j *subCols + x] = 32767;
				}
			}
		}
		hipMalloc((void**) &b, sizeof(short) * subCols * k);
		hipMemcpy(b, temp3, sizeof(short)*subCols*k, hipMemcpyHostToDevice);

		free(temp3);

		for(unsigned long y = 0; y < numSubMatrixA + 1; ++y){
			if(overflowA == 0 && y == numSubMatrixA){
				break;
			}

            short *a;
			short *temp = (short*) malloc( sizeof(short)*subRows * k );
			for(unsigned long j = 0; j < subRows; ++j){
				for(unsigned long x = 0; x < k; ++x){
					if(y * subRows + j < m){
						temp[j * k + x] = A[y*subRows*k + j*k + x];
					}else{
						temp[j * k + x] = 32767;
					}
				}			
			}
			hipMalloc((void**) &a, sizeof(short) * subRows * k);
                        hipMemcpy(a, temp, sizeof(short)*subRows*k, hipMemcpyHostToDevice);

                        short* c;
			hipMalloc((void**) &c, sizeof(short) * subCols * subRows);
			doMultiply2Matrices_jam(subRows, k, a, k, subCols, b, c); 			
			hipMemcpy(temp, c, sizeof(short)*subRows*subCols, hipMemcpyDeviceToHost);

			if(i == numSubMatrixB && y == numSubMatrixA){
				copyElements(C, temp, subRows, subCols, m, n, y, i, overflowA, overflowB);
			}else if(i == numSubMatrixB){
				copyElements(C, temp, subRows, subCols, m, n, y, i, 0, overflowB);
			}else if(y == numSubMatrixA){
				copyElements(C, temp, subRows, subCols, m, n, y, i, overflowA, 0);
			}else{
				copyElements(C, temp, subRows, subCols, m, n, y, i, 0, 0);
			}
			free(temp);
			hipFree(a);
			hipFree(c);
		
		}
		
		hipFree(b);
	}
}





