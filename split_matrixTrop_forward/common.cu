#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "common.h"

#define BLOCK_SIZE 32

#define MIN(a,b) \
   ({ __typeof__ (a) _a = (a); \
       __typeof__ (b) _b = (b); \
     _a > _b ? _b : _a; })

__global__ void gpu_matrix_mult(short *a,
		                short *b, 
				short *c, 
				unsigned long m, 
				unsigned long n, 
				unsigned long k)
{ 
    unsigned long row = blockIdx.y * blockDim.y + threadIdx.y; 
    unsigned long col = blockIdx.x * blockDim.x + threadIdx.x;

    if( col < k && row < m) 
    {
        short sum = 32767;
        for(unsigned long i = 0; i < n; i++) 
        {
            sum = MIN( sum, (a[row * n + i] + b[i * k + col]) );
        }
        c[row * k + col] = sum;
    }
} 


void copyElements(short* out, 
                  short* entry, 
                  unsigned long eRows, 
                  unsigned long eCols, 
                  unsigned long oRows, 
                  unsigned long oCols, 
                  unsigned long x, 
                  unsigned long y,
	          unsigned long ofA, 
                  unsigned long ofB){

	unsigned long counterRows = eRows;
	unsigned long counterCols = eCols;
	if(ofA){
		counterRows = ofA;
	}
	if(ofB){
		counterCols = ofB;	
	}
	for(unsigned long i = 0; i < counterRows; ++i){
		for(unsigned long j = 0; j < counterCols; ++j){
			unsigned long index = x*eRows*oCols + (i*oCols) + (y*eCols + j);
                        out[index] = entry[i*eCols + j];
		}

	}
}


void doMultiply2Matrices_jam(
        unsigned long  a1Rows, unsigned long a1Cols,  short* A1,
        unsigned long a2Rows, unsigned long a2Cols,  short* A2,
	short* C)
{

	//int devID = 0;
        hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp,0);
        hipSetDevice(0);


	unsigned long grid_rows = (a1Rows + BLOCK_SIZE - 1) / BLOCK_SIZE;
        unsigned long grid_cols = (a2Rows + BLOCK_SIZE - 1) / BLOCK_SIZE;
        dim3 dimGrid(grid_cols, grid_rows);
        dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);

        gpu_matrix_mult<<< dimGrid, dimBlock >>>(A1, A2, C, a1Rows, a2Rows, a2Cols);

	hipDeviceSynchronize();

}

