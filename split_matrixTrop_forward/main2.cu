#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include <iostream>

#include "kernel1.cu"

#include "TimingCPU.h"
#include "TimingGPU.cuh"

#define MIN(a, b) \
    ({ __typeof__ (a) _a = (a); \
       __typeof__ (b) _b = (b); \
     _a > _b ? _b : _a; })

#define CHECK_CUDA_ERROR(val) check((val), #val, __FILE__, __LINE__)
template <typename T>
void check(T err, const char *const func, const char *const file,
           const int line)
{
    if (err != hipSuccess)
    {
        std::cerr << "CUDA Runtime Error at: " << file << ":" << line
                  << std::endl;
        std::cerr << hipGetErrorString(err) << " " << func << std::endl;
        // We don't exit when we encounter CUDA errors in this example.
        // std::exit(EXIT_FAILURE);
    }
}

// Array with matrix power
short **h_C;

// Vector with min of main diagonal
short *min_Diagonal;

short bm;
int am;
short bm2;
int n0m;

short calculo_min_diagonales_2(long matCrow, long matCcol, int indice_matriz);
int matrixMultiply(int block_size, unsigned long matArow, unsigned long matAcol, unsigned long matBrow, unsigned long matBcol, int iteraciones);
int loadfile(dim3 &dimsA, dim3 &dimsB, char *filename1, int iteraciones);
int print_min_diagonales(unsigned long matArow, unsigned long matAcol, unsigned long matBrow, unsigned long matBcol);
int busqueda_recurrencia_CPU(unsigned long matCrow, unsigned long matCcol, int iteraciones);


int loadfile(dim3 &dimsA, dim3 &dimsB, char *filename1, int iteraciones)
{

    FILE *f1;

    //f1 = fopen(filename1, "r");
  
    if ( (f1 = fopen(filename1, "rb")) == NULL)
     {
         fprintf(stderr, "Error opening file.");
         exit(1);
     }

    // Allocate host matrix C

    dim3 dimsC(dimsB.x, dimsA.y, 1);

    //Array de matrices
    //Solo asignamos memoria para la primera entrada
    size_t size_C = (unsigned long)(dimsC.x) * (unsigned long)(dimsC.y);
    h_C = (short **)malloc(sizeof(short *) * (iteraciones + 1));
    
    h_C[0] = (short *)malloc(size_C * sizeof(short));

    if (fread(h_C[0],sizeof(short),size_C, f1) != size_C)
     {
         fprintf(stderr, "Error reading file.");
         exit(1);
     }
     
     //printf("Hola ... %d %d %d %d %d %d\n",h_C[50][0],h_C[50][1],h_C[50][2],h_C[50][3],h_C[50][4],h_C[50][5]);

    fclose(f1);


    printf("Archivo cargado en memoria.....\n");

    // Allocate vector for minimal of main diagonal

    min_Diagonal = (short *)malloc(sizeof(short) * (iteraciones + 1));
    min_Diagonal[0] = calculo_min_diagonales_2((long)dimsC.x, (long)dimsC.y, 0);

    return 0;
}

/**
 *  Run matrix_trop and create de array with power
 */
int matrixMultiply(int block_size, unsigned long matArow, unsigned long matAcol, unsigned long matBrow, unsigned long matBcol, int iteraciones)
{


    unsigned long  A_sz, B_sz, C_sz;
    short *A_h, *B_h, *C_h;
    
    A_sz = matArow*matAcol;
    B_sz = matBrow*matBcol;
    C_sz = matArow*matBcol;

    printf("\nMatrix size : %ld\n",A_sz*2); fflush(stdout);
    printf("Numero de submatrices : %d\n",num_submatrix);
    printf("Numero de iteraciones : %d\n",iteraciones);

    A_h = (short*) malloc( sizeof(short)*A_sz );
    B_h = (short*) malloc( sizeof(short)*B_sz );
    C_h = (short*) malloc( sizeof(short)*C_sz );

    //unsigned long jj;

    if (A_h == NULL)
    {
        printf("Memory not allocated.\n");
        exit(0);
    }

    if (B_h == NULL)
    {
        printf("Memory not allocated.\n");
        exit(0);
    }

    if (C_h == NULL)
    {
        printf("Memory not allocated.\n");
        exit(0);
    }


    memcpy(A_h,h_C[0],A_sz*sizeof(short));
    memcpy(B_h,h_C[0],A_sz*sizeof(short));
    
/*
FILE *f2;
if ( (f2 = fopen("13_8_50.bin", "rb")) == NULL)
     {
         fprintf(stderr, "Error opening file.");
         exit(1);
     }
if (fread(B_h,sizeof(short),B_sz, f2) != B_sz)
     {
         fprintf(stderr, "Error reading file.");
         exit(1);
     }
fclose(f2);
memcpy(h_C[50],B_h,C_sz*sizeof(short));


printf("Hola ... %d %d %d %d %d %d\n",h_C[50][0],h_C[50][1],h_C[50][2],h_C[50][3],h_C[50][4],h_C[50][5]);
printf("Hola ... %d %d %d %d %d %d\n",A_h[0],A_h[1],A_h[2],A_h[3],A_h[4],A_h[5]);
printf("Hola ... %d %d %d %d %d %d\n",B_h[0],B_h[1],B_h[2],B_h[3],B_h[4],B_h[5]);
*/

    int devID = 0;
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, devID);
    printf("\nDevice %d: \"%s\"\n\n", devID, deviceProp.name);
    hipSetDevice(0);
   
    // Execute the kernel

    int recurrencia;


    for (unsigned int it = 1; it < (iteraciones + 1); it++)
    {

        printf("Entrando al msplit %hu..... ",it);fflush(stdout);
        
        msplitm(matArow, matBcol, matBrow, A_h, matArow, B_h, matBrow, C_h, matBrow);

        hipDeviceSynchronize();

        printf(".. saliendo de msplit\n");

        //Asignamos memoria para la matriz a guardar
        h_C[it] = (short *)malloc(C_sz * sizeof(short));

        memcpy(B_h,C_h,B_sz*sizeof(short));
        memcpy(h_C[it],C_h,B_sz*sizeof(short));

        /*Guardamos una salida cada multiplo de 1a*/
	/*
	if((it%25)==0){
         char filename2[20];
	 FILE *f2;
         sprintf(filename2,"%lu_%d_%ld.txt",13,num_submatrix,it);
         f2=fopen(filename2,"w");

	 if (fwrite(h_C[it],sizeof(short),C_sz, f2) != C_sz)
          {
            fprintf(stderr, "Error reading file.");
            exit(1);
          }
	 fclose(f2);
	}
        */

	printf("Entrada al calculo de las diagonales ....");fflush(stdout);
        min_Diagonal[it] = calculo_min_diagonales_2(matAcol, matArow, it);
	printf(".. saliendo del calculo de las diagonales\n");
	
        recurrencia = 1;
        if (it > 7) //Entro a partir de la potencia 7
        {
	    printf("Entrando al calculo de la recurrencia....");fflush(stdout);
            recurrencia = busqueda_recurrencia_CPU(matAcol, matArow, it);
            if (recurrencia == 1){
	      printf(".. saliendo por continue...no se ha encontrado\n");
	      continue; 
            }else{
	      printf(".. saliendo por break...se ha encontrado!!!\n");
	      break;
            }
	}
        

    }

    // Liberamos la memoria
    // Menos las potencias calculadas que estan en h_C
    free(A_h);
    free(B_h);
    free(C_h);

    return 0;
}

int busqueda_recurrencia_CPU(unsigned long matCrow, unsigned long matCcol, int iteracion)
{

    size_t size_C = (unsigned long)(matCrow) * (unsigned long)(matCcol);
    short *temp = (short *)malloc(size_C * sizeof(short));

    int it2;
    int salida;

    // hago las diferencias
    for (it2 = (iteracion-1); it2 > (iteracion-7); it2--)
    {

        for (unsigned long j = 0; j < size_C; j++)
            if (h_C[it2][j] == 32767) temp[j]= 32767;
	    else temp[j] = h_C[iteracion][j] - h_C[it2][j];

        bm = temp[0];
        salida = 0;
        for (unsigned long i = 1; i < size_C; i++)
        {
            
            if (temp[i] >= 32767)
            {
                salida = 0;
                continue;
            }
            
            if (bm != temp[i])
            {
                salida = 1;
                break;
            }
        }

        if (salida == 0)
        {
            break;
        }

    } // del bucle it2

    am = iteracion - it2;
    n0m = it2 + 1;

    free(temp);
    free(h_C[iteracion-7]);

    return salida;
}

int print_min_diagonales(unsigned long matArow, unsigned long matAcol, unsigned long matBrow, unsigned long matBcol, int hasta)
{

    for (unsigned long i = 0; i < (hasta + 1); i++)
    {
        printf("Potencia %ld | Menor %d \n", i, min_Diagonal[i - 1]);
    }
    return 0;
}

short calculo_min_diagonales_2(long matCrow, long matCcol, int indice_matriz)
{
    short menor;
    size_t size_C = (unsigned long)(matCrow) * (unsigned long)(matCcol);

    // Recorremos la diagonal buscando el minimo
    menor = 32767;
    for (unsigned long j = 0; j < size_C; j = (j + matCrow + 1))
        if (h_C[indice_matriz][j] < menor)
            menor = h_C[indice_matriz][j];

    return menor;
}

int main(int argc, char **argv)
{

    TimingCPU timer_CPU;
    TimingGPU timer_GPU;

    char *filename1 = argv[1];

    unsigned long matArow, matAcol;
    unsigned long matBrow, matBcol;

    matArow = atoi(argv[2]);
    matAcol = atoi(argv[2]);
    matBrow = atoi(argv[2]);
    matBcol = atoi(argv[2]);

    int block_size = 32;

    int iteraciones = atoi(argv[3]);

    num_submatrix = atoi(argv[4]);

    dim3 dimsA(1, 1, 1);
    dim3 dimsB(1, 1, 1);
    dimsA.x = matAcol;
    dimsA.y = matArow;
    dimsB.x = matBcol;
    dimsB.y = matBrow;

    timer_CPU.StartCounter();
    int load_result = loadfile(dimsA, dimsB, filename1, iteraciones);
    float t_upload_file = timer_CPU.GetCounter() / 1000.0;

    timer_GPU.StartCounter();
    int matrix_result = matrixMultiply(block_size, matArow, matAcol, matBrow, matBcol, iteraciones);
    float t_calculate_recurrence = timer_CPU.GetCounter() / 1000.0;

    int diagonales = print_min_diagonales(matArow, matAcol, matBrow, matBcol, (n0m + am - 1));

/*
    for (unsigned int i = 0; i < (iteraciones + 1); i++)
        free(h_C[i]);
*/
    free(h_C);

    printf("====================\n");
    printf("n0m=%d am=%d bm=%d\n", n0m, am, (int)bm);
    printf("====================\n");
    printf("Time :\n");
    printf("Upload file          : %.5f seg\n", t_upload_file);
    printf("Calculate recurrence : %.5f seg\n", t_calculate_recurrence);
    printf("====================\n");

    exit(0);
}
