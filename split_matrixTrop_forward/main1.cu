#include <stdio.h>
#include <stdlib.h>
#include "kernel1.cu"

// --- Timing includes
#include "TimingCPU.h"
#include "TimingGPU.cuh"

int main (int argc, char *argv[])
{

    TimingCPU timer_CPU,timer1_CPU;
    TimingGPU timer_GPU;

    // Initialize host variables ----------------------------------------------

    printf("\nSetting up the problem..."); fflush(stdout);

    short *A_h, *B_h, *C_h;
    unsigned long  A_sz, B_sz, C_sz;
    unsigned long matArow, matAcol;
    unsigned long matBrow, matBcol;
    char *filename1;
    char filename2[20];
    FILE *f1;
    FILE *f2;


    unsigned long iteraciones = atoi(argv[4]);
    unsigned long tamano = atoi(argv[5]);
    num_submatrix = atoi(argv[2]);
    matArow = atoi(argv[3]);
    matAcol = matBrow = atoi(argv[3]);
    matBcol = atoi(argv[3]);
    filename1=argv[1];

    timer_CPU.StartCounter();

    A_sz = matArow*matAcol;
    B_sz = matBrow*matBcol;
    C_sz = matArow*matBcol;

    printf("\nMatrix size : %ld\n",A_sz); fflush(stdout);

    A_h = (short*) malloc( sizeof(short)*A_sz );
    B_h = (short*) malloc( sizeof(short)*B_sz );
    C_h = (short*) malloc( sizeof(short)*C_sz );

    printf("Opening file : %s\n",filename1);
    f1=fopen(filename1,"r");
    if (f1==NULL) {printf("File error..\n"); fflush(stdout); exit (1);}

    short a;
    unsigned long ii;
    unsigned long jj;
    int count;

    for (ii = 0; ii < A_sz; ii++){
        count=fscanf(f1,"%hu",&a);
        if (count == 0)
        {
            printf("Error loading file..\n");
            exit(0);
        }
        A_h[ii] = a;
        B_h[ii] = a;
        //C_h[ii] = 9999;
    }

    fclose(f1);

    float t_upload_file=timer_CPU.GetCounter()/1000.0;

    printf("A: %lu x %lu \tB: %lu x %lu\tC: %lu x %lu\n", 
        matArow, 
	matAcol, 
	matBrow, 
	matBcol, 
	matArow, 
	matBcol);


    // Launch kernel using msplitm ---------------------------

    printf("Launching kernel...\n"); fflush(stdout);

    timer_GPU.StartCounter();

    for(unsigned long i = 0; i < iteraciones; i++){

     msplitm(matArow, matBcol, matBrow, A_h, matArow, B_h, matBrow, C_h, matBrow);
     hipDeviceSynchronize();

     //OJO Quitado solo es la medida pura del producto de matrices
     //Tenemos que copiar la matriz C en B y volver a iterar
     
     printf("Hemos salido de msplitm\n");

     for (jj=0; jj < B_sz; jj++) { B_h[jj] = C_h[jj]; }

    //}

    float t_iteration=timer_GPU.GetCounter()/1000.0;

    timer1_CPU.StartCounter();

    // Save matrix C in file
    sprintf(filename2,"%lu_%d_%ld.txt",tamano,num_submatrix,i);
    f2=fopen(filename2,"w");
    for (jj = 0; jj < C_sz; jj++){
                    fprintf(f2,"%d\n",C_h[jj]);
    }

    float t_download_file=timer1_CPU.GetCounter()/1000.0;


    }

    // Free memory ------------------------------------------------------------

    free(A_h);
    free(B_h);
    free(C_h);

    //printf("Upload data    : %.6f seg.\n", t_upload_file);
    //printf("Iteration time : %.6f seg.\n", t_iteration);
    //printf("Download file  : %.6f seg.\n", t_download_file);
    //printf("==========================\n");
    //printf("Total          : %.6f seg.\n", t_upload_file+t_iteration+t_download_file);

    return 0;

}

